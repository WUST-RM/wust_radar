#include <iostream>
#include "NvidiaInterface.hpp"
#include "classify.hpp"

namespace classify {

using namespace std;

int postprocess(vector<float>& output_array)
{
    int   max_index = 0;
    float max_value = output_array[0];
    for (int i = 1; i < output_array.size(); i++) {
        if (output_array[i] > max_value) {
            max_value = output_array[i];
            max_index = i;
        }
    }
    return max_index;
}

inline int upbound(int n, int align = 32)
{
    return (n + align - 1) / align * align;
}

class InferImpl : public Infer<int> {
public:
    shared_ptr<trt::Infer>                         trt_;
    string                                         engine_file_;
    Type                                           type_;
    vector<shared_ptr<trt::Memory<unsigned char>>> preprocess_buffers_;
    trt::Memory<float> input_buffer_, output_array_;
    int                num_class_ = 0;
    int                network_input_width_, network_input_height_;
    bool               isdynamic_model_ = false;
    Norm               normalize_;
    float              mean_[3], std_[3];

    virtual ~InferImpl() = default;

    void adjust_memory(int batch_size)
    {
        size_t input_numel =
            network_input_width_ * network_input_height_ * 3;
        input_buffer_.gpu(batch_size * input_numel);
        output_array_.gpu(batch_size * num_class_);
        output_array_.cpu(batch_size * num_class_);

        if ((int)preprocess_buffers_.size() < batch_size) {
            for (int i = preprocess_buffers_.size(); i < batch_size; ++i)
                preprocess_buffers_.push_back(
                    make_shared<trt::Memory<unsigned char>>());
        }
    }

    void
    preprocess(int ibatch, const Image& image,
               shared_ptr<trt::Memory<unsigned char>> preprocess_buffer,
               AffineMatrix& affine, void* stream = nullptr)
    {
        affine.compute(
            make_tuple(image.width, image.height),
            make_tuple(network_input_width_, network_input_height_));

        size_t input_numel =
            network_input_width_ * network_input_height_ * 3;
        float*   input_device = input_buffer_.gpu() + ibatch * input_numel;
        size_t   size_image = image.width * image.height * 3;
        size_t   size_matrix = upbound(sizeof(affine.d2i), 32);
        uint8_t* gpu_workspace =
            preprocess_buffer->gpu(size_matrix + size_image);
        float*   affine_matrix_device = (float*)gpu_workspace;
        uint8_t* image_device = gpu_workspace + size_matrix;

        uint8_t* cpu_workspace =
            preprocess_buffer->cpu(size_matrix + size_image);
        float*   affine_matrix_host = (float*)cpu_workspace;
        uint8_t* image_host = cpu_workspace + size_matrix;

        hipStream_t stream_ = (hipStream_t)stream;
        std::chrono ::high_resolution_clock::time_point a1 =
            std::chrono::high_resolution_clock::now();
        memcpy(image_host, image.bgrptr, size_image);
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        std::chrono ::high_resolution_clock::time_point a2 =
            std::chrono::high_resolution_clock::now();
        auto time_used2 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a2 -
                                                                      a1);
        checkRuntime(hipMemcpyAsync(image_device, image_host, size_image,
                                     hipMemcpyHostToDevice, stream_));
        checkRuntime(hipMemcpyAsync(affine_matrix_device,
                                     affine_matrix_host, sizeof(affine.d2i),
                                     hipMemcpyHostToDevice, stream_));
        warp_affine_bilinear_and_normalize_plane(
            image_device, image.width * 3, image.width, image.height,
            input_device, network_input_width_, network_input_height_,
            affine_matrix_device, 114, normalize_, stream_);
    }

    bool load(const string& engine_file, Type type)
    {
        trt_ = trt::load(engine_file);
        if (trt_ == nullptr)
            return false;

        trt_->print();

        this->type_ = type;

        auto input_dim = trt_->static_dims(0);
        auto output_dims_ = trt_->static_dims(1);
        puts("input_dim");
        for (int i = 0; i < 4; i++) {
            std::cout << input_dim[i] << std::endl;
        }

        network_input_width_ = input_dim[3];
        network_input_height_ = input_dim[2];
        isdynamic_model_ = trt_->has_dynamic_dim();

        mean_[0] = 0.485;
        mean_[1] = 0.456;
        mean_[2] = 0.406;
        std_[0] = 0.229;
        std_[1] = 0.224;
        std_[2] = 0.225;
        normalize_ =
            Norm::mean_std(mean_, std_, 1.0 / 255.0, ChannelType::SwapRB);
        num_class_ = output_dims_[1];

        return true;
    }

    virtual int forward(const Image& image, void* stream = nullptr) override
    {
        auto output = forwards({image}, stream);
        if (output.empty())
            return {};
        return output[0];
    }

    virtual vector<int> forwards(const vector<Image>& images,
                                 void* stream = nullptr) override
    {
        int num_image = images.size();
        if (num_image == 0)
            return {};

        auto input_dims = trt_->static_dims(0);
        int  infer_batch_size = input_dims[0];
        if (infer_batch_size != num_image) {
            if (isdynamic_model_) {
                infer_batch_size = num_image;
                input_dims[0] = num_image;
                if (!trt_->set_run_dims(0, input_dims))
                    return {};
            } else {
                if (infer_batch_size < num_image) {
                    INFO("When using static shape model, number of "
                         "images[%d] must be "
                         "less than or equal to the maximum batch[%d].",
                         num_image, infer_batch_size);
                    return {};
                }
            }
        }

        std::chrono ::high_resolution_clock::time_point a1 =
            std::chrono::high_resolution_clock::now();
        adjust_memory(infer_batch_size);  // 调用内存
        std::chrono ::high_resolution_clock::time_point a2 =
            std::chrono::high_resolution_clock::now();
        auto time_used2 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a2 -
                                                                      a1);

        vector<AffineMatrix> affine_matrixs(num_image);
        std::chrono ::high_resolution_clock::time_point a3 =
            std::chrono::high_resolution_clock::now();
        hipStream_t stream_ = (hipStream_t)stream;
        for (int i = 0; i < num_image; ++i)
            preprocess(i, images[i], preprocess_buffers_[i],
                       affine_matrixs[i], stream);

        vector<void*> bindings{input_buffer_.gpu(), output_array_.gpu()};

        std::chrono ::high_resolution_clock::time_point a3d1 =
            std::chrono::high_resolution_clock::now();
        auto time_used3 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a3d1 -
                                                                      a3);
        if (!trt_->forward(bindings, stream)) {
            INFO("Failed to tensorRT forward.");
            return {};
        }
        std::chrono ::high_resolution_clock::time_point a3d2 =
            std::chrono::high_resolution_clock::now();
        auto time_used3d2 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a3d2 -
                                                                      a3d1);
        std::chrono ::high_resolution_clock::time_point a3d3 =
            std::chrono::high_resolution_clock::now();
        auto time_used3d3 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a3d3 -
                                                                      a3d2);

        std::chrono ::high_resolution_clock::time_point a3d4 =
            std::chrono::high_resolution_clock::now();
        checkRuntime(hipMemcpyAsync(
            output_array_.cpu(), output_array_.gpu(),
            output_array_.gpu_bytes(), hipMemcpyDeviceToHost, stream_));
        checkRuntime(hipStreamSynchronize(stream_));
        std::chrono ::high_resolution_clock::time_point a4 =
            std::chrono::high_resolution_clock::now();
        auto time_used3d4 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a4 -
                                                                      a3d4);

        auto time_used4 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a4 -
                                                                      a3);
        // INFO("forward and decode_kernel_invoker time: %f",
        //      time_used4.count() * 1000);
        std::chrono ::high_resolution_clock::time_point a5 =
            std::chrono::high_resolution_clock::now();
        vector<int> arrout(num_image);
        for (int ib = 0; ib < num_image; ++ib) {
            float*        parray = output_array_.cpu() + ib * num_class_;
            vector<float> output(num_class_);
            for (int o = 0; o < num_class_; o++) {
                output[o] = parray[o];
            }

            arrout[ib] = postprocess(output);
        }

        return arrout;
    }
};
Infer<int>* loadraw(const std::string& engine_file, Type type)
{
    InferImpl* impl = new InferImpl();
    if (!impl->load(engine_file, type)) {
        delete impl;
        impl = nullptr;
    }
    return impl;
}
shared_ptr<Infer<int>> load(const string& engine_file, Type type)
{
    return std::shared_ptr<InferImpl>(
        (InferImpl*)loadraw(engine_file, type));
}
}  // namespace classify
